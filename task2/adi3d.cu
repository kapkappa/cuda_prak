#include "hip/hip_runtime.h"
#include "cudadefs.h"

#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

namespace gpu {

template <uint32_t BLOCKSIZE>
__device__ __forceinline__ void warp_reduce(size_t i, volatile double* data) {
    if (BLOCKSIZE >= 64) data[i] += data[i + 32];
    if (BLOCKSIZE >= 32) data[i] += data[i + 16];
    if (BLOCKSIZE >= 16) data[i] += data[i +  8];
    if (BLOCKSIZE >=  8) data[i] += data[i +  4];
    if (BLOCKSIZE >=  4) data[i] += data[i +  2];
    if (BLOCKSIZE >=  2) data[i] += data[i +  1];
}

template <uint32_t BLOCKSIZE>
__device__ __forceinline__ void warp_reduce_max(size_t i, volatile double* data) {
    if (BLOCKSIZE >= 64) data[i] = max(data[i], data[i + 32]);
    if (BLOCKSIZE >= 32) data[i] = max(data[i], data[i + 16]);
    if (BLOCKSIZE >= 16) data[i] = max(data[i], data[i +  8]);
    if (BLOCKSIZE >=  8) data[i] = max(data[i], data[i +  4]);
    if (BLOCKSIZE >=  4) data[i] = max(data[i], data[i +  2]);
    if (BLOCKSIZE >=  2) data[i] = max(data[i], data[i +  1]);
}

template <uint32_t BLOCKSIZE>
__global__ void get_eps(const double * __restrict__ A, size_t NX, size_t NY, size_t NZ, double *eps_out) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id
    const size_t idz = blockIdx.z * blockDim.z + threadIdx.z; // Z-axis thread id

    const size_t id = idx + idy * NX + idz * NX * NY;

    const size_t thread_id = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;  // thread index in block
    const size_t block_id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;         // block index in grid

    __shared__ double shared_eps[BLOCKSIZE];    //1-dimensional shared memory

    double tmp = 0.0;

    if (0 < idx && idx < (NX-1) && 0 < idy && idy < (NY-1) && 0 < idz && idz < (NZ-1)) {
        tmp = A[id + NX*NY] - A[id - NX*NY];
        tmp = fabs(A[id] - tmp);
    }

    shared_eps[thread_id] = tmp;

    __syncthreads();

//  Unroll block-wise reduction
//    if (BLOCKSIZE >= 512) { if (thread_id < 256) { shared_eps[thread_id] += shared_eps[thread_id + 256]; } __syncthreads(); }
//    if (BLOCKSIZE >= 256) { if (thread_id < 128) { shared_eps[thread_id] += shared_eps[thread_id + 128]; } __syncthreads(); }
//    if (BLOCKSIZE >= 128) { if (thread_id <  64) { shared_eps[thread_id] += shared_eps[thread_id +  64]; } __syncthreads(); }
    if (BLOCKSIZE >= 512) { if (thread_id < 256) { shared_eps[thread_id] = max(shared_eps[thread_id], shared_eps[thread_id + 256]); } __syncthreads(); }
    if (BLOCKSIZE >= 256) { if (thread_id < 128) { shared_eps[thread_id] = max(shared_eps[thread_id], shared_eps[thread_id + 128]); } __syncthreads(); }
    if (BLOCKSIZE >= 128) { if (thread_id <  64) { shared_eps[thread_id] = max(shared_eps[thread_id], shared_eps[thread_id +  64]); } __syncthreads(); }

    if (thread_id < 32) { warp_reduce_max<BLOCKSIZE>(thread_id, shared_eps); }

    if (thread_id == 0) {
        eps_out[block_id] = shared_eps[0];
    }

    return;
}

__global__ void update(double * A, size_t NX, size_t NY, size_t NZ, size_t offset) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id
    const size_t idz = blockIdx.z * blockDim.z + threadIdx.z; // Z-axis thread id

    const size_t id = idx + idy * NX + idz * NX * NY;

    if (idx == 0 || idx >= (NX-1) || idy == 0 || idy >= (NY-1) || idz == 0 || idz >= (NZ-1)) {
        return;
    }

    A[id] = (A[id - offset] + A[id - offset]) / 2.0;

    return;
}

__global__ void update1(double * A, size_t NX, size_t NY, size_t NZ, size_t offset) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id
    const size_t idz = blockIdx.z * blockDim.z + threadIdx.z; // Z-axis thread id

    const size_t id = idx + idy * NX + idz * NX * NY;

    if (idx == 0 || idx >= (NX-1) || idy == 0 || idy >= (NY-1) || idz == 0 || idz >= (NZ-1)) {
        return;
    }

    A[id] = (A[id - offset] + A[id - offset]) / 2.0;

    return;
}

__global__ void update2(double * A, size_t NX, size_t NY, size_t NZ, size_t offset) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id
    const size_t idz = blockIdx.z * blockDim.z + threadIdx.z; // Z-axis thread id

    const size_t id = idx + idy * NX + idz * NX * NY;

    if (idx == 0 || idx >= (NX-1) || idy == 0 || idy >= (NY-1) || idz == 0 || idz >= (NZ-1)) {
        return;
    }

    A[id] = (A[id - offset] + A[id - offset]) / 2.0;

    return;
}

__global__ void update3(double * A, size_t NX, size_t NY, size_t NZ, size_t offset) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id

    const size_t id = idx + idy * NX;

    if (idx == 0 || idx >= (NX-1) || idy == 0 || idy >= (NY-1)) {
        return;
    }

    for (size_t k = 1; k < NZ-1; k++) {
        size_t idz = k * offset;
        A[id + k] = (A[id + idz + offset] + A[id + idz - offset]) / 2.0;
    }
    return;
}


double update_wrapper(double *A, double *B, size_t NX, size_t NY, size_t NZ) {

    dim3 threads_per_block = dim3(X_BLOCKSIZE, Y_BLOCKSIZE, Z_BLOCKSIZE);
    dim3 blocks_per_grid = dim3((NX-1) / threads_per_block.x + 1,
                                (NY-1) / threads_per_block.y + 1,
                                (NZ-1) / threads_per_block.z + 1);

    double offset_x = 1;
    double offset_y = NY;
    double offset_z = NX * NY;

    update1<<<1, 1>>>(A, NX, NY, NZ, offset_x);

    int threads_per_block_2 = 256;
    dim3 blocks_per_grid_2 = dim3((NX-1) / threads_per_block_2);
    update2<<<blocks_per_grid_2, threads_per_block_2>>>(A, NX, NY, NZ, offset_y);

    uint32_t grid_size = blocks_per_grid.x * blocks_per_grid.y * blocks_per_grid.z;
    thrust::device_vector<double> eps_out(grid_size);
    get_eps<TOTAL_BLOCKSIZE><<<blocks_per_grid, threads_per_block>>>(A, NX, NY, NZ,  thrust::raw_pointer_cast(eps_out.data()));
    thrust::device_ptr<double> eps_ptr = eps_out.data();
    double eps = *(thrust::max_element(eps_ptr, eps_ptr + grid_size));

    dim3 threads_per_block_3 = dim3(X_BLOCKSIZE, Y_BLOCKSIZE);
    dim3 blocks_per_grid_3 = dim3((NX-1) / threads_per_block_3.x,
                                  (NY-1) / threads_per_block_3.y);
    update3<<<blocks_per_grid_3, threads_per_block_3>>>(A, NX, NY, NZ, offset_z);

    return eps;
}


} // gpu
