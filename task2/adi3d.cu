#include "hip/hip_runtime.h"
#include "cudadefs.h"

#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

namespace gpu {

#define Max(a, b) ((a) > (b) ? (a) : (b))

template <uint32_t BLOCKSIZE>
__device__ __forceinline__ void warp_reduce_max(size_t i, volatile double* data) {
    if (BLOCKSIZE >= 64) { data[i] = Max(data[i], data[i + 32]); }
    if (BLOCKSIZE >= 32) { data[i] = Max(data[i], data[i + 16]); }
    if (BLOCKSIZE >= 16) { data[i] = Max(data[i], data[i +  8]); }
    if (BLOCKSIZE >=  8) { data[i] = Max(data[i], data[i +  4]); }
    if (BLOCKSIZE >=  4) { data[i] = Max(data[i], data[i +  2]); }
    if (BLOCKSIZE >=  2) { data[i] = Max(data[i], data[i +  1]); }
}

__global__ void update1(double * A, size_t NX, size_t NY, size_t NZ, size_t idx) {

    const size_t idy = blockIdx.x * blockDim.x + threadIdx.x; // Y-axis thread id
    const size_t idz = blockIdx.y * blockDim.y + threadIdx.y; // Z-axis thread id

    if (idy == 0 || idy >= NX-1 || idz == 0 || idz >= NY-1) {
        return;
    }

    const size_t id = idx + idy * NX + idz * NX * NY;
    A[id] = (A[id + 1] + A[id - 1]) / 2.0;

    return;
}

__global__ void update2(double * A, size_t NX, size_t NY, size_t NZ, size_t idy) {

    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idz = blockIdx.y * blockDim.y + threadIdx.y; // Z-axis thread id

    if (idx == 0 || idx >= NX-1 || idz == 0 || idz >= NZ-1) {
        return;
    }

    const size_t id = idx + idy * NX + idz * NX * NY;
    A[id] = (A[id + NX] + A[id - NX]) / 2.0;

    return;
}

template <uint32_t BLOCKSIZE>
__global__ void get_eps(double * A, size_t NX, size_t NY, size_t NZ, double *eps_out, size_t idz) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id

    const size_t id = idx + idy * NX + idz * NX * NY;

    const size_t thread_id = threadIdx.x + threadIdx.y * blockDim.x;  // thread index in block
    const size_t block_id = blockIdx.x + blockIdx.y * gridDim.x;         // block index in grid

    __shared__ double shared_eps[BLOCKSIZE];    //1-dimensional shared memory

    double tmp = 0.0;

    if (0 < idx && idx < (NX-1) && 0 < idy && idy < (NY-1)) {
        double tmp2 = (A[id + NX*NY] + A[id - NX*NY]) / 2.0;
        tmp = fabs(A[id] - tmp2);
        A[id] = tmp2;
    }

    shared_eps[thread_id] = tmp;

    __syncthreads();

//  Unroll block-wise reduction
    if (BLOCKSIZE >= 1024) { if (thread_id < 512) { shared_eps[thread_id] = Max(shared_eps[thread_id], shared_eps[thread_id + 512]); } __syncthreads(); }
    if (BLOCKSIZE >= 512 ) { if (thread_id < 256) { shared_eps[thread_id] = Max(shared_eps[thread_id], shared_eps[thread_id + 256]); } __syncthreads(); }
    if (BLOCKSIZE >= 256 ) { if (thread_id < 128) { shared_eps[thread_id] = Max(shared_eps[thread_id], shared_eps[thread_id + 128]); } __syncthreads(); }
    if (BLOCKSIZE >= 128 ) { if (thread_id <  64) { shared_eps[thread_id] = Max(shared_eps[thread_id], shared_eps[thread_id +  64]); } __syncthreads(); }

    if (thread_id < 32) { warp_reduce_max<BLOCKSIZE>(thread_id, shared_eps); }

    if (thread_id == 0) {
        eps_out[block_id] = shared_eps[0];
    }

    return;
}


double update_wrapper(double *A, size_t NX, size_t NY, size_t NZ, dim3 BPG, dim3 TPB, double * eps_out) {

    // 3.63 sec
    for (int i = 1; i < NX-1; i++) {
        update1<<<BPG, TPB>>>(A, NX, NY, NZ, i);
    }

    // 0.33 sec
    for (int j = 1; j < NY-1; j++) {
        update2<<<BPG, TPB>>>(A, NX, NY, NZ, j);
    }

    double eps = 0.0;

    uint32_t grid_size = BPG.x * BPG.y;

    thrust::device_ptr<double> eps_ptr = thrust::device_pointer_cast(eps_out);

    for (int k = 1; k < NZ-1; k++) {
        get_eps<TOTAL_BLOCKSIZE><<<BPG, TPB>>>(A, NX, NY, NZ, eps_out, k);  // 0.45

        double local_eps = *(thrust::max_element(eps_ptr, eps_ptr + grid_size));    // 1.35

        eps = Max(eps, local_eps);
    }

    return eps;
}


} // gpu
