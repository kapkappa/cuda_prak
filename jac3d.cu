#include "hip/hip_runtime.h"
/* Jacobi-3 program */

#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>

#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

#define Max(a, b) ((a) > (b) ? (a) : (b))

#define MAXEPS 0.5

#define X_BLOCKSIZE 8
#define Y_BLOCKSIZE 8
#define Z_BLOCKSIZE 8
#define TOTAL_BLOCKSIZE (X_BLOCKSIZE * Y_BLOCKSIZE * Z_BLOCKSIZE)

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

static inline double timer() {
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp, &tzp);
    return((double)tp.tv_sec + (double)tp.tv_usec * 1.e-06);
}

static inline size_t get_index(size_t i, size_t j, size_t k, size_t size) {
    return i * size * size + j * size + k;
}

template <uint32_t BLOCKSIZE>
__device__ __forceinline__ void block_reduce_max(size_t i, double* data) {
    if (BLOCKSIZE > 512) { if (i < 512 && i + 512 < BLOCKSIZE) { data[i] = Max(data[i], data[i + 512]); } __syncthreads(); }
    if (BLOCKSIZE > 256) { if (i < 256 && i + 256 < BLOCKSIZE) { data[i] = Max(data[i], data[i + 256]); } __syncthreads(); }
    if (BLOCKSIZE > 128) { if (i < 128 && i + 128 < BLOCKSIZE) { data[i] = Max(data[i], data[i + 128]); } __syncthreads(); }
    if (BLOCKSIZE >  64) { if (i <  64 && i +  64 < BLOCKSIZE) { data[i] = Max(data[i], data[i +  64]); } __syncthreads(); }
    if (BLOCKSIZE >  32) { if (i <  32 && i +  32 < BLOCKSIZE) { data[i] = Max(data[i], data[i +  32]); } __syncthreads(); }
    if (BLOCKSIZE >  16) { if (i <  16 && i +  16 < BLOCKSIZE) { data[i] = Max(data[i], data[i +  16]); } __syncthreads(); }
    if (BLOCKSIZE >   8) { if (i <   8 && i +   8 < BLOCKSIZE) { data[i] = Max(data[i], data[i +   8]); } __syncthreads(); }
    if (BLOCKSIZE >   4) { if (i <   4 && i +   4 < BLOCKSIZE) { data[i] = Max(data[i], data[i +   4]); } __syncthreads(); }
    if (BLOCKSIZE >   2) { if (i <   2 && i +   2 < BLOCKSIZE) { data[i] = Max(data[i], data[i +   2]); } __syncthreads(); }
    if (BLOCKSIZE >   1) { if (i <   1 && i +   1 < BLOCKSIZE) { data[i] = Max(data[i], data[i +   1]); } __syncthreads(); }
}

template <uint32_t BLOCKSIZE>
__global__ void jacobi(double *A, double *B, size_t NX, size_t NY, size_t NZ, double *eps_out) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id
    const size_t idz = blockIdx.z * blockDim.z + threadIdx.z; // Z-axis thread id

    const size_t id = idx + idy * NX + idz * NX * NY;

    const size_t thread_id = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;  // thread index in block
    const size_t block_id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;         // block index in grid

    __shared__ double shared_eps[BLOCKSIZE];    //1-dimensional shared memory

    double tmp = 0.0;

    if (0 < idx && idx < (NX-1) && 0 < idy && idy < (NY-1) && 0 < idz && idz < (NZ-1)) {
        tmp = fabs(B[id] - A[id]);
        A[id] = B[id];
    }

    shared_eps[thread_id] = tmp;

    __syncthreads();

    block_reduce_max<BLOCKSIZE>(thread_id, shared_eps);

    if (thread_id == 0) {
        eps_out[block_id] = shared_eps[0];
    }

    if (idx == 0 || idx >= (NX-1) || idy == 0 || idy >= (NY-1) || idz == 0 || idz >= (NZ-1)) {
        return;
    }

    size_t offset_x = 1;
    size_t offset_y = NX;
    size_t offset_z = NX * NY;

    B[id] = (A[id - offset_x] + A[id - offset_y] + A[id - offset_z] + A[id + offset_x] + A[id + offset_y] + A[id + offset_z]) / 6.0;
}

double solve(double *A, double *B, size_t size) {
    size_t NX = size, NY = size, NZ = size;
    double eps = 0.0;

    for (size_t i = 1; i < NX-1; i++) {
        for (size_t j = 1; j < NY-1; j++) {
            for (size_t k = 1; k < NZ-1; k++) {
                size_t idx = get_index(i, j, k, size);
                double tmp = fabs(B[idx] - A[idx]);
                eps = Max(tmp, eps);
                A[idx] = B[idx];
            }
        }
    }

    size_t offset_i = NZ * NY;
    size_t offset_j = NZ;
    size_t offset_k = 1;

    for (size_t i = 1; i < NX-1; i++) {
        for (size_t j = 1; j < NY-1; j++) {
            for (size_t k = 1; k < NZ-1; k++) {
                size_t idx = get_index(i, j, k, size);
                B[idx] = (A[idx - offset_i] + A[idx - offset_j] + A[idx - offset_k] +
                          A[idx + offset_i] + A[idx + offset_j] + A[idx + offset_k]) / 6.0;
            }
        }
    }

    return eps;
}

int main(int argc, char **argv) {

    int argc_indx = 0;
    int iters = 100;
    size_t size = 30;
    std::string driver = "CPU";
    enum class driver_t {CPU, GPU} drv = driver_t::CPU;
    while (argc_indx < argc) {
        if (!strcmp(argv[argc_indx], "-size")) {
            argc_indx++;
            size = atoi(argv[argc_indx]);
        } else if (!strcmp(argv[argc_indx], "-iters")) {
            argc_indx++;
            iters = atoi(argv[argc_indx]);
        } else if (!strcmp(argv[argc_indx], "-driver")) {
            argc_indx++;
            if (!strcmp(argv[argc_indx], "GPU")) {
                drv = driver_t::GPU;
                driver = "GPU";
            } else if (!strcmp(argv[argc_indx], "CPU")) {
                drv = driver_t::CPU;
                driver = "CPU";
            } else {
                printf("Wrong driver! Set to CPU.\n");
            }
        } else if (!strcmp(argv[argc_indx], "-help")) {
            printf("Usage: ./prog_gpu -size L -iters N\n");
            return 0;
        } else {
            argc_indx++;
        }
    }

    size_t NX = size, NY = size, NZ = size;

    double *h_A, *h_B;

    if ((h_A = (double*)malloc(sizeof(double) * NX * NY * NZ)) == NULL) { perror("matrix host_A allocation failed"); exit(1); }
    if ((h_B = (double*)malloc(sizeof(double) * NX * NY * NZ)) == NULL) { perror("matrix host_B allocation failed"); exit(1); }

    // Init
    for (size_t i = 0; i < NX; i++) {
        for (size_t j = 0; j < NY; j++) {
            for (size_t k = 0; k < NZ; k++) {
                size_t idx = get_index(i, j, k, size);
                h_A[idx] = 0;
                if (i == 0 || j == 0 || k == 0 || i == NX-1 || j == NY-1 || k == NZ-1) {
                    h_B[idx] = 0.0;
                } else {
                    h_B[idx] = 4.0 + i + j + k;
                }
            }
        }
    }

    double *d_A, *d_B;
    CHECK_CUDA( hipMalloc(&d_A, NX * NY * NZ * sizeof(double)) )
    CHECK_CUDA( hipMalloc(&d_B, NX * NY * NZ * sizeof(double)) )

    CHECK_CUDA( hipMemcpy(d_A, h_A, sizeof(double) * NX * NY * NZ, hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(d_B, h_B, sizeof(double) * NX * NY * NZ, hipMemcpyHostToDevice) )

    dim3 threads_per_block = dim3(X_BLOCKSIZE, Y_BLOCKSIZE, Z_BLOCKSIZE);
    dim3 blocks_per_grid = dim3((size-1) / threads_per_block.x + 1,
                                (size-1) / threads_per_block.y + 1,
                                (size-1) / threads_per_block.z + 1);

//    constexpr int block_size = threads_per_block.x * threads_per_block.y * threads_per_block.z;
    uint32_t grid_size = blocks_per_grid.x * blocks_per_grid.y * blocks_per_grid.z;

    double eps = 0.0, *eps_out;
    hipMalloc(&eps_out, sizeof(double) * grid_size);

    double t1 = timer();

    for (int it = 1; it <= iters; it++) {

        if (drv == driver_t::GPU) {

            jacobi<TOTAL_BLOCKSIZE><<<blocks_per_grid, threads_per_block>>>(d_A, d_B, NX, NY, NZ, eps_out);

            thrust::device_ptr<double> eps_ptr = thrust::device_pointer_cast(eps_out);
            eps = *(thrust::max_element(eps_ptr, eps_ptr + grid_size));

        } else {
            eps = solve(h_A, h_B, size);
        }

        printf(" IT = %4i   EPS = %14.12E\n", it, eps);
        if (eps < MAXEPS)
            break;
    }

    double t2 = timer();

    free(h_A);
    free(h_B);

    CHECK_CUDA( hipFree(d_A) )
    CHECK_CUDA( hipFree(d_B) )
    CHECK_CUDA( hipFree(eps_out) )

    printf(" Jacobi3D Benchmark Completed.\n");
    printf(" Size            = %4ld x %4ld x %4ld\n", NX, NY, NZ);
    printf(" Iterations      =       %12d\n", iters);
    printf(" Time in seconds =       %12.6lf\n", t2 - t1);
    printf(" Operation type  =     floating point\n");
    printf(" Driver          = %s\n", driver.c_str());
//    printf(" Verification    =       %12s\n", (fabs(eps - 5.058044) < 1e-11 ? "SUCCESSFUL" : "UNSUCCESSFUL"));
    printf(" END OF Jacobi3D Benchmark\n");
    return 0;
}
