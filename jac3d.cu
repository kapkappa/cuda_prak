#include "hip/hip_runtime.h"
/* Jacobi-3 program */

#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>

#include <thrust/extrema.h>
#include <thrust/device_ptr.h>


#define Max(a, b) ((a) > (b) ? (a) : (b))

#define MAXEPS 0.5f

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

static inline double timer() {
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp, &tzp);
    return((double)tp.tv_sec + (double)tp.tv_usec * 1.e-06);
}

template <uint32_t BLOCKSIZE>
__device__ __forceinline__ void block_reduce_max(size_t i, double* data) {
    if (BLOCKSIZE > 512) { if (i < 512 && i + 512 < BLOCKSIZE) { data[i] = Max(data[i], data[i + 512]); } __syncthreads(); }
    if (BLOCKSIZE > 256) { if (i < 256 && i + 256 < BLOCKSIZE) { data[i] = Max(data[i], data[i + 256]); } __syncthreads(); }
    if (BLOCKSIZE > 128) { if (i < 128 && i + 128 < BLOCKSIZE) { data[i] = Max(data[i], data[i + 128]); } __syncthreads(); }
    if (BLOCKSIZE >  64) { if (i <  64 && i +  64 < BLOCKSIZE) { data[i] = Max(data[i], data[i +  64]); } __syncthreads(); }
    if (BLOCKSIZE >  32) { if (i <  32 && i +  32 < BLOCKSIZE) { data[i] = Max(data[i], data[i +  32]); } __syncthreads(); }
    if (BLOCKSIZE >  16) { if (i <  16 && i +  16 < BLOCKSIZE) { data[i] = Max(data[i], data[i +  16]); } __syncthreads(); }
    if (BLOCKSIZE >   8) { if (i <   8 && i +   8 < BLOCKSIZE) { data[i] = Max(data[i], data[i +   8]); } __syncthreads(); }
    if (BLOCKSIZE >   4) { if (i <   4 && i +   4 < BLOCKSIZE) { data[i] = Max(data[i], data[i +   4]); } __syncthreads(); }
    if (BLOCKSIZE >   2) { if (i <   2 && i +   2 < BLOCKSIZE) { data[i] = Max(data[i], data[i +   2]); } __syncthreads(); }
    if (BLOCKSIZE >   1) { if (i <   1 && i +   1 < BLOCKSIZE) { data[i] = Max(data[i], data[i +   1]); } __syncthreads(); }
}

template <uint32_t BLOCKSIZE>
__global__ void jacobi(double *A, double *B, size_t NX, size_t NY, size_t NZ, double *eps_out) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id
    const size_t idz = blockIdx.z * blockDim.z + threadIdx.z; // Z-axis thread id

    const size_t id = idx + idy * NX + idz * NX * NY;

    const size_t thread_id = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;  // thread index in block
    const size_t block_id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;         // block index in grid

    __shared__ double shared_eps[BLOCKSIZE];    //1-dimensional shared memory

    double tmp = 0.0;

    if (0 < idx && idx < (NX-1) && 0 < idy && idy < (NY-1) && 0 < idz && idz < (NZ-1)) {
        tmp = fabs(B[id] - A[id]);
        A[id] = B[id];
    }

    shared_eps[thread_id] = tmp;

    __syncthreads();

    block_reduce_max<BLOCKSIZE>(thread_id, shared_eps);

    if (thread_id == 0) {
        eps_out[block_id] = shared_eps[0];
    }

    if (idx == 0 || idx >= (NX-1) || idy == 0 || idy >= (NY-1) || idz == 0 || idz >= (NZ-1)) {
        return;
    }

    size_t offset_x = 1;
    size_t offset_y = NX;
    size_t offset_z = NX * NY;

    B[id] = (A[id - offset_x] + A[id - offset_y] + A[id - offset_z] + A[id + offset_x] + A[id + offset_y] + A[id + offset_z]) / 6.0;
}



int main(int argc, char **argv) {

    int argc_indx = 0;
    int iters = 100;
    size_t size = 30;
    while (argc_indx < argc) {
        if (!strcmp(argv[argc_indx], "-size")) {
            argc_indx++;
            size = atoi(argv[argc_indx]);
        } else if (!strcmp(argv[argc_indx], "-iters")) {
            argc_indx++;
            iters = atoi(argv[argc_indx]);
        } else if (!strcmp(argv[argc_indx], "-help")) {
            printf("Usage: ./prog_gpu -size L -iters N\n");
            return 0;
        } else {
            argc_indx++;
        }
    }

    size_t NX = size, NY = size, NZ = size;

    double *h_A, *h_B;

    if ((h_A = (double*)malloc(sizeof(double) * NX * NY * NZ)) == NULL) { perror("matrix host_A allocation failed"); exit(1); }
    if ((h_B = (double*)malloc(sizeof(double) * NX * NY * NZ)) == NULL) { perror("matrix host_B allocation failed"); exit(1); }

    // Init
    for (size_t i = 0; i < NX; i++) {
        for (size_t j = 0; j < NY; j++) {
            for (size_t k = 0; k < NZ; k++) {
                h_A[i * NY * NZ + j * NZ + k] = 0;
                if (i == 0 || j == 0 || k == 0 || i == NX-1 || j == NY-1 || k == NZ-1) {
                    h_B[i * NY * NZ + j * NZ + k] = 0.0;
                } else {
                    h_B[i * NY * NZ + j * NZ + k] = 4.0 + i + j + k;
                }
            }
        }
    }

    double *d_A, *d_B;
    CHECK_CUDA( hipMalloc(&d_A, NX * NY * NZ * sizeof(double)) )
    CHECK_CUDA( hipMalloc(&d_B, NX * NY * NZ * sizeof(double)) )

    CHECK_CUDA( hipMemcpy(d_A, h_A, sizeof(double) * NX * NY * NZ, hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(d_B, h_B, sizeof(double) * NX * NY * NZ, hipMemcpyHostToDevice) )

    dim3 threads_per_block = dim3(8, 8, 8);
    dim3 blocks_per_grid = dim3((size-1) / threads_per_block.x + 1,
                                (size-1) / threads_per_block.y + 1,
                                (size-1) / threads_per_block.z + 1);

//    constexpr int block_size = threads_per_block.x * threads_per_block.y * threads_per_block.z;
    uint32_t grid_size = blocks_per_grid.x * blocks_per_grid.y * blocks_per_grid.z;

    double eps = 0.0, *eps_out;
    hipMalloc(&eps_out, sizeof(double) * grid_size);

    double t1 = timer();

    for (int it = 1; it <= iters; it++) {
        eps = 0.0;

        jacobi<512><<<blocks_per_grid, threads_per_block>>>(d_A, d_B, NX, NY, NZ, eps_out);

/*
        hipMemcpy(eps_host, eps_out, sizeof(double) * grid_size, hipMemcpyDeviceToHost);
        for (uint32_t i = 0; i < grid_size; i++) {
            eps = Max(eps, eps_host[i]);
            eps_host[i] = 0.0;
        }
        hipMemcpy(eps_out, eps_host, sizeof(double) * grid_size, hipMemcpyHostToDevice);
*/
        thrust::device_ptr<double> eps_ptr = thrust::device_pointer_cast(eps_out);
        eps = *(thrust::max_element(eps_ptr, eps_ptr + grid_size));

        printf(" IT = %4i   EPS = %14.12E\n", it, eps);
        if (eps < MAXEPS)
            break;
    }

    double t2 = timer();

    free(h_A);
    free(h_B);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(eps_out);

    printf(" Jacobi3D Benchmark Completed.\n");
    printf(" Size            = %4ld x %4ld x %4ld\n", NX, NY, NZ);
    printf(" Iterations      =       %12d\n", iters);
    printf(" Time in seconds =       %12.6lf\n", t2 - t1);
    printf(" Operation type  =     floating point\n");
//    printf(" Verification    =       %12s\n", (fabs(eps - 5.058044) < 1e-11 ? "SUCCESSFUL" : "UNSUCCESSFUL"));

    printf(" END OF Jacobi3D Benchmark\n");
    return 0;
}
