#include "hip/hip_runtime.h"
/* Jacobi-3 program */

#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>

#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

#define Max(a, b) ((a) > (b) ? (a) : (b))

#define MAX_EPS 5E-1
#define MAX_DIFF 1E-6

#ifndef X_BLOCKSIZE
#define X_BLOCKSIZE 4
#endif
#ifndef Y_BLOCKSIZE
#define Y_BLOCKSIZE 2
#endif
#ifndef Z_BLOCKSIZE
#define Z_BLOCKSIZE 4
#endif

#define TOTAL_BLOCKSIZE (X_BLOCKSIZE * Y_BLOCKSIZE * Z_BLOCKSIZE)

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

static inline double timer() {
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp, &tzp);
    return((double)tp.tv_sec + (double)tp.tv_usec * 1.e-06);
}

// For square matrices only!
static inline size_t get_index(size_t i, size_t j, size_t k, size_t size) {
    return i * size * size + j * size + k;
}


/////////////////////////////////////////////////////////////////////


template <uint32_t BLOCKSIZE>
__device__ __forceinline__ void warp_reduce(size_t i, volatile double* data) {
    if (BLOCKSIZE >= 64) data[i] += data[i + 32];
    if (BLOCKSIZE >= 32) data[i] += data[i + 16];
    if (BLOCKSIZE >= 16) data[i] += data[i +  8];
    if (BLOCKSIZE >=  8) data[i] += data[i +  4];
    if (BLOCKSIZE >=  4) data[i] += data[i +  2];
    if (BLOCKSIZE >=  2) data[i] += data[i +  1];
}

template <uint32_t BLOCKSIZE>
__global__ void get_eps(hipPitchedPtr d_A, hipPitchedPtr d_B, size_t NX, size_t NY, size_t NZ, double *eps_out) {

    char * A = (char*)d_A.ptr;
    char * B = (char*)d_B.ptr;

    size_t step_y = d_A.pitch;
	size_t step_z = d_A.pitch * NX;

    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id
    const size_t idz = blockIdx.z * blockDim.z + threadIdx.z; // Z-axis thread id

    const size_t thread_id = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;  // thread index in block
    const size_t block_id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;         // block index in grid

    __shared__ double shared_eps[BLOCKSIZE];    //1-dimensional shared memory

    double tmp = 0.0;

    if (0 < idx && idx < (NX-1) && 0 < idy && idy < (NY-1) && 0 < idz && idz < (NZ-1)) {
        tmp = ((double*)(B + idy * step_y + idz * step_z))[idx] - ((double*)(A + idy * step_y + idz * step_z))[idx];
    }

    shared_eps[thread_id] = tmp * tmp;

    __syncthreads();

//  Unroll block-wise reduction
    if (BLOCKSIZE >= 512) { if (thread_id < 256) { shared_eps[thread_id] += shared_eps[thread_id + 256]; } __syncthreads(); }
    if (BLOCKSIZE >= 256) { if (thread_id < 128) { shared_eps[thread_id] += shared_eps[thread_id + 128]; } __syncthreads(); }
    if (BLOCKSIZE >= 128) { if (thread_id <  64) { shared_eps[thread_id] += shared_eps[thread_id +  64]; } __syncthreads(); }

    if (thread_id < 32) { warp_reduce<BLOCKSIZE>(thread_id, shared_eps); }

    if (thread_id == 0) {
        eps_out[block_id] = shared_eps[0];
    }

    return;
}

__global__ void update(hipPitchedPtr d_A, hipPitchedPtr d_B, size_t NX, size_t NY, size_t NZ) {

    char * A = (char *)d_A.ptr;
    char * B = (char *)d_B.ptr;

    size_t step_y = d_A.pitch;
	size_t step_z = d_A.pitch * NX;

    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // X-axis thread id
    const size_t idy = blockIdx.y * blockDim.y + threadIdx.y; // Y-axis thread id
    const size_t idz = blockIdx.z * blockDim.z + threadIdx.z; // Z-axis thread id

    char* A_id = A + idy * step_y + idz * step_z;
    char* B_id = B + idy * step_y + idz * step_z;

    if (idx == 0 || idx >= (NX-1) || idy == 0 || idy >= (NY-1) || idz == 0 || idz >= (NZ-1)) {
        return;
    }

    ((double*)B_id)[idx] = ( ((double*)(A_id))[idx + 1] + ((double*)(A_id))[idx - 1] +
                             ((double*)(A_id + step_y))[idx] + ((double*)(A_id - step_y))[idx] +
                             ((double*)(A_id + step_z))[idx] + ((double*)(A_id - step_z))[idx] ) / 6.0;
}


/////////////////////////////////////////////////////////////////////


double get_eps(const double *__restrict__ A, const double *__restrict__ B, size_t size) {
    size_t NX = size, NY = size, NZ = size;
    double eps = 0.0;

    for (size_t i = 1; i < NX-1; i++) {
        for (size_t j = 1; j < NY-1; j++) {
            for (size_t k = 1; k < NZ-1; k++) {
                size_t idx = get_index(i, j, k, size);
                double tmp = B[idx] - A[idx];
                eps += tmp * tmp;
            }
        }
    }
    return sqrt(eps);
}

void jac3d(double *A, double *B, size_t size) {
    size_t NX = size, NY = size, NZ = size;

    for (size_t i = 1; i < NX-1; i++) {
        for (size_t j = 1; j < NY-1; j++) {
            for (size_t k = 1; k < NZ-1; k++) {
                size_t idx = get_index(i, j, k, size);
                A[idx] = B[idx];
            }
        }
    }

    size_t offset_i = NZ * NY;
    size_t offset_j = NZ;
    size_t offset_k = 1;

    for (size_t i = 1; i < NX-1; i++) {
        for (size_t j = 1; j < NY-1; j++) {
            for (size_t k = 1; k < NZ-1; k++) {
                size_t idx = get_index(i, j, k, size);
                B[idx] = (A[idx - offset_k] + A[idx - offset_j] + A[idx - offset_i] +
                          A[idx + offset_k] + A[idx + offset_j] + A[idx + offset_i]) / 6.0;
            }
        }
    }
}


/////////////////////////////////////////////////////////////////////


int main(int argc, char **argv) {

    int argc_indx = 0;
    int iters = 100;
    size_t size = 30;
    std::string driver = "CPU";
    bool verification = false;
    enum class driver_t {CPU, GPU} drv = driver_t::CPU;
    while (argc_indx < argc) {
        if (!strcmp(argv[argc_indx], "-size")) {
            argc_indx++;
            size = atoi(argv[argc_indx]);
        } else if (!strcmp(argv[argc_indx], "-iters")) {
            argc_indx++;
            iters = atoi(argv[argc_indx]);
        } else if (!strcmp(argv[argc_indx], "-driver")) {
            argc_indx++;
            if (!strcmp(argv[argc_indx], "GPU")) {
                drv = driver_t::GPU;
                driver = "GPU";
            } else if (!strcmp(argv[argc_indx], "CPU")) {
                drv = driver_t::CPU;
                driver = "CPU";
            } else {
                printf("Wrong driver! Set to CPU.\n");
            }
        } else if (!strcmp(argv[argc_indx], "-help")) {
            printf("Usage: ./prog_gpu -size L -iters N -driver [CPU|GPU] [-verification]\n");
            return 0;
        } else if (!strcmp(argv[argc_indx], "-verification")) {
            argc_indx++;
            verification = true;
        } else {
            argc_indx++;
        }
    }

    size_t NX = size, NY = size, NZ = size;

    double *h_A, *h_B;

    if ((h_A = (double*)malloc(sizeof(double) * NX * NY * NZ)) == NULL) { perror("matrix host_A allocation failed"); exit(1); }
    if ((h_B = (double*)malloc(sizeof(double) * NX * NY * NZ)) == NULL) { perror("matrix host_B allocation failed"); exit(2); }

    // Init
    for (size_t i = 0; i < NX; i++) {
        for (size_t j = 0; j < NY; j++) {
            for (size_t k = 0; k < NZ; k++) {
                size_t idx = get_index(i, j, k, size);
                h_A[idx] = 0;
                if (i == 0 || j == 0 || k == 0 || i == NX-1 || j == NY-1 || k == NZ-1) {
                    h_B[idx] = 0.0;
                } else {
                    h_B[idx] = 4.0 + i + j + k;
                }
            }
        }
    }

    hipExtent extent_bytes = make_hipExtent(sizeof(double) * NX, NY, NZ);

    hipPitchedPtr dev_pitched_A, dev_pitched_B;

    CHECK_CUDA( hipMalloc3D(&dev_pitched_A, extent_bytes) )  // Allocate pitched structure
    CHECK_CUDA( hipMalloc3D(&dev_pitched_B, extent_bytes) )

    CHECK_CUDA( hipMemset3D(dev_pitched_A, 0.0, extent_bytes) )  // Init dev_A with 0.0
    CHECK_CUDA( hipMemset3D(dev_pitched_B, 0.0, extent_bytes) )

    hipPitchedPtr host_pitched_A = make_hipPitchedPtr((void*)h_A, sizeof(double) * NX, NY, NZ);
    hipPitchedPtr host_pitched_B = make_hipPitchedPtr((void*)h_B, sizeof(double) * NX, NY, NZ);

    hipMemcpy3DParms params = {0};
    params.extent = extent_bytes;
    params.kind   = hipMemcpyHostToDevice;

    params.srcPtr = host_pitched_A;
    params.dstPtr = dev_pitched_A;

    hipMemcpy3D(&params);

    params.srcPtr = host_pitched_B;
    params.dstPtr = dev_pitched_B;

    hipMemcpy3D(&params);


    dim3 threads_per_block = dim3(X_BLOCKSIZE, Y_BLOCKSIZE, Z_BLOCKSIZE);
    dim3 blocks_per_grid = dim3((size-1) / threads_per_block.x + 1,
                                (size-1) / threads_per_block.y + 1,
                                (size-1) / threads_per_block.z + 1);

    uint32_t grid_size = blocks_per_grid.x * blocks_per_grid.y * blocks_per_grid.z;

    double eps = 1.0, *eps_out;
    CHECK_CUDA( hipMalloc(&eps_out, sizeof(double) * grid_size) )

    int it = 0;
    double t1 = 0.0, t2 = 0.0, t3 = 0.0;
    float time1 = 0.0, time2 = 0.0;

    double *cpu_eps = NULL, *gpu_eps = NULL;
    if (verification) {
        if ((cpu_eps = (double*)calloc(iters, sizeof(double))) == NULL) { perror("cpu_eps allocation failed"); exit(3); }
        if ((gpu_eps = (double*)calloc(iters, sizeof(double))) == NULL) { perror("gpu_eps allocation failed"); exit(4); }
    }


    if (verification || (drv == driver_t::CPU)) {
        t1 = timer();

        for (it = 0; it < iters; it++) {
            jac3d(h_A, h_B, size);

            if (verification) {
                eps = get_eps(h_A, h_B, size);
                cpu_eps[it] = eps;
            }
        }

        t2 = timer();
        time1 = t2-t1;

        if (!verification) {
            eps = get_eps(h_A, h_B, size);
        }

        t3 = timer();
        time2 = t3-t2;
    }


    if (verification || (drv == driver_t::GPU)) {

        hipEvent_t start, stop;

        CHECK_CUDA( hipEventCreate(&start) )
        CHECK_CUDA( hipEventCreate(&stop) )

        CHECK_CUDA( hipEventRecord(start, 0) )

        params.dstPtr = dev_pitched_A;
        params.srcPtr = dev_pitched_B;

        for (it = 0; it < iters; it++) {
            hipMemcpy3D(&params);
            update<<<blocks_per_grid, threads_per_block>>>(dev_pitched_A, dev_pitched_B, NX, NY, NZ);

            if (verification) {
                get_eps<TOTAL_BLOCKSIZE><<<blocks_per_grid, threads_per_block>>>(dev_pitched_A, dev_pitched_B, NX, NY, NZ, eps_out);
                thrust::device_ptr<double> eps_ptr = thrust::device_pointer_cast(eps_out);
                eps = sqrt( thrust::reduce(thrust::device, eps_ptr, eps_ptr + grid_size, 0.0) );
                gpu_eps[it] = eps;
            }
        }

        CHECK_CUDA( hipEventRecord(stop, 0) )

        CHECK_CUDA( hipEventSynchronize(stop) )
        CHECK_CUDA( hipEventElapsedTime(&time1, start, stop) )

        time1 = time1 / 1000.0;

        CHECK_CUDA( hipEventRecord(start, 0) )

        if (!verification) {
            get_eps<TOTAL_BLOCKSIZE><<<blocks_per_grid, threads_per_block>>>(dev_pitched_A, dev_pitched_B, NX, NY, NZ, eps_out);
            thrust::device_ptr<double> eps_ptr = thrust::device_pointer_cast((double *)eps_out);
            eps = sqrt( thrust::reduce(thrust::device, eps_ptr, eps_ptr + grid_size, 0.0) );
        }

        CHECK_CUDA( hipEventRecord(stop, 0) )
        CHECK_CUDA( hipEventSynchronize(stop) )
        CHECK_CUDA( hipEventElapsedTime(&time2, start, stop) )

        time2 = time2 / 1000.0;

        CHECK_CUDA( hipEventDestroy(start) )
        CHECK_CUDA( hipEventDestroy(stop) )
    }


    if (verification) {
        for (int i = 0; i < it; i++) {
            double tmp = fabs(cpu_eps[i] - gpu_eps[i]);
            if (tmp >= MAX_DIFF) {
                printf(" IT = %4i, EPS check failed!\n", i);
                printf("cpu_eps[%i] = %3.11E gpu_eps[%i] = %3.11E, diff = %3.11E\n", i, cpu_eps[i], i, gpu_eps[i], tmp);
            }
        }
        free(cpu_eps);
        free(gpu_eps);
    }

    free(h_A);
    free(h_B);

    CHECK_CUDA( hipFree(dev_pitched_A.ptr) )
    CHECK_CUDA( hipFree(dev_pitched_B.ptr) )
    CHECK_CUDA( hipFree(eps_out) )

    if (verification) {
        printf("\n ===================================\n");
        printf(" Verification Completed.\n");
        printf(" Final eps      = %1.12E\n", eps);
        printf(" Test size      = %4ld x %4ld x %4ld\n", NX, NY, NZ);
        printf(" Test iters     =       %12d\n", iters);
        printf(" Operation type =     floating point\n");
        printf("\n ===================================\n");
    } else {
        printf("\n ===================================\n");
        printf(" Jacobi3D Benchmark Completed.\n");
        printf(" Final eps       = %1.12E\n", eps);
        printf(" Size            = %4ld x %4ld x %4ld\n", NX, NY, NZ);
        printf(" Iterations      =       %12d\n", it);
        printf(" Jacobi Time     =       %8.6lf sec\n", time1);
        printf(" 1 Eps Time      =       %8.6lf sec\n", time2);
        printf(" Operation type  =     floating point\n");
        printf(" Driver          = %18s\n", driver.c_str());
        printf(" END OF Jacobi3D Benchmark\n");
        printf("\n ===================================\n");
    }
    return 0;
}
